#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define _USE_MATH_DEFINES // for C
#include <math.h>

#include <time.h> // Added to use the clock() function
#include <hip/hip_runtime.h>

#include "AC2D.h"
#include "model.h"
#include "rec.h"
#include "src.h"

double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return (double)ts.tv_sec + (double)ts.tv_nsec / 1000000000.0;
}


int main(){
    
    // FD solver
    AC2D* ac2d = NULL;  
    
    // Model
    Model* model = NULL; 
    int Nx=1001, Ny=1001; // Model dimension in x- and y-directions.
    int Nb=15; // Border for PML attenuation
    double dt=0.0005, dx=10; // Time sampling and space sampling intervals
    double f0=25.0, t0=0.04; // Peak frequency, Pulse delay
    int l=8; // Operator length

    double W0=f0*3.1415*2.0; // Central angular frequency
    double **vp = NULL, **rho = NULL, **Q = NULL;
    double vp0=2000, rho0=2000, q0=10e5;  
    int rheol = 0;
    
    
    // Variable for receiver 
    Rec* rec = NULL;   
    int nt=1000; // No of time steps
    int resamp=1, sresamp=1; // Resampling factors for data and snapshot; Output receiver sampling, Output snapshot resampling  
    
    // Variables for source
    Src* src = NULL;   
    double ricker_wavelet[nt]; // Source pulse
    int sx, sy; // Source x,y-coordinates
    int Nr=100; // Number of receivers, number of receivers X
    int rx[Nr], ry[Nr];     // Receiver x,y-coordinates
    
    // Variable declaration for main
    int i, j; // Timestep no
    //char* tmp; // Temporary workspace
    
    double si;
    
    vp = (double**)malloc(Nx * sizeof(double*));
    rho = (double**)malloc(Nx * sizeof(double*));
    Q = (double**)malloc(Nx * sizeof(double*));
    for (int i = 0; i < Nx; i++) {
        vp[i] = (double*)malloc(Ny * sizeof(double));
        rho[i] = (double*)malloc(Ny * sizeof(double));
        Q[i] = (double*)malloc(Ny * sizeof(double));
        
        for(j=0; j < Ny; j++){
            /*if (j < Ny/2) {
                vp[i][j] = 2000;
                rho[i][j] = 2000;
                Q[i][j] = 10e5;
            }
            else {
                vp[i][j] = 3000;
                rho[i][j] = 3000;
                Q[i][j] = 10e5;
            }
            */
            vp[i][j] = vp0; /* Read the velocity model */
            rho[i][j] = rho0; /* Read the density model */
            Q[i][j] = q0; /* Read the attenuation model */
        }
    }
    
    /* Create a source */
    sx= Nx/2;
    sy= Ny/2;
    SrcRicker(ricker_wavelet, t0, f0, nt, dt);
    src=SrcNew(ricker_wavelet,sx,sy);

    // Print ricker_wavelet
    /*printf("Ricker wavelet:\n");
    for (int i = 0; i < nt; i++) {
        printf("%f ", ricker_wavelet[i]);
    }
    printf("\n");*/
    
    /* Create a model */
    rheol = MAXWELL;
    
    model = ModelNew(vp,rho,Q,dx,dt,W0,Nb, rheol, Nx, Ny);
    

    si = ModelStability(model);

    printf("Stability index: %f\n", si);
    

    /* Create a receiver */
    for(i=0; i<Nr; i=i+1){
        rx[i] = 200;
        ry[i] = i;
    }
    
    
    rec=RecNew(rx,ry,nt,resamp,sresamp,Nx,Ny,dx,Nb,dt,Nr);
    
    /* Create solver */
    ac2d = Ac2dNew(model);

    // Add timer before Ac2dSolve
    double start_time = get_time();

    /* Run solver */
    Ac2dSolve(ac2d, model, src, rec, nt,l);

    // Add timer after Ac2dSolve and print the time
    hipDeviceSynchronize();
    double end_time = get_time();

    double elapsed= end_time - start_time;
    printf("Solver wall clock time: %f seconds\n", elapsed);


    /* Save recording */
    save_wavefield(rec);
    printf("OK : \n");

    /* Free memory */
    // Free memory when you're done
    for (int i = 0; i < Nx; i++) {
        free(vp[i]);
        free(rho[i]);
        free(Q[i]);
    }
    free(vp);
    free(rho);
    free(Q);
    SrcDel(src);
    ac2dDel(ac2d, Nx);
    ModelDel(model, Nx);
    return OK;
}

