#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


__global__ void Ac2dvx_kernel(double* vx_flat, double* Rho_flat, double* exx_flat, double* thetax_flat, double* Drhox_flat, double* Eta1x_flat, double* Eta2x_flat, int Nx, int Ny, double Dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < Nx && j < Ny) {
        vx_flat[i * Ny + j] = Dt * (1.0 / Rho_flat[i * Ny + j]) * exx_flat[i * Ny + j] + vx_flat[i * Ny + j] + Dt * thetax_flat[i * Ny + j] * Drhox_flat[i * Ny + j];
        thetax_flat[i * Ny + j] = Eta1x_flat[i * Ny + j] * thetax_flat[i * Ny + j] + Eta2x_flat[i * Ny + j] * exx_flat[i * Ny + j];
    }
}

__global__ void Ac2dvy_kernel(double* vy_flat, double* Rho_flat, double* eyy_flat, double* thetay_flat, double* Drhoy_flat, double* Eta1y_flat, double* Eta2y_flat, int Nx, int Ny, double Dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < Nx && j < Ny) {
        vy_flat[i * Ny + j] = Dt * (1.0 / Rho_flat[i * Ny + j]) * eyy_flat[i * Ny + j] + vy_flat[i * Ny + j] + Dt * thetay_flat[i * Ny + j] * Drhoy_flat[i * Ny + j];
        thetay_flat[i * Ny + j] = Eta1y_flat[i * Ny + j] * thetay_flat[i * Ny + j] + Eta2y_flat[i * Ny + j] * eyy_flat[i * Ny + j];
    }
}

__global__ void Ac2dstress_kernel(double* p_flat, double* Kappa_flat, double* exx_flat, double* eyy_flat, double* gammax_flat, double* gammay_flat, double* Dkappax_flat, double* Dkappay_flat, double* Alpha1x_flat, double* Alpha2x_flat, double* Alpha1y_flat, double* Alpha2y_flat, int Nx, int Ny, double Dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < Nx && j < Ny) {
        p_flat[i * Ny + j] = Dt * Kappa_flat[i * Ny + j] * (exx_flat[i * Ny + j] + eyy_flat[i * Ny + j]) + p_flat[i * Ny + j] + Dt * (gammax_flat[i * Ny + j] * Dkappax_flat[i * Ny + j] + gammay_flat[i * Ny + j] * Dkappay_flat[i * Ny + j]);
        gammax_flat[i * Ny + j] = Alpha1x_flat[i * Ny + j] * gammax_flat[i * Ny + j] + Alpha2x_flat[i * Ny + j] * exx_flat[i * Ny + j];
        gammay_flat[i * Ny + j] = Alpha1y_flat[i * Ny + j] * gammay_flat[i * Ny + j] + Alpha2y_flat[i * Ny + j] * eyy_flat[i * Ny + j];
    }
}


__global__ void DiffDxplus_kernel(double* A, double* dA, double* w, double dx, int Nx, int Ny, int l) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    double sum;
    int k;

    if (i < Nx && j < Ny) {
        // Top border (1 < i < l+1)
        if (i < l) {
            sum = 0.0;
            for (k = 1; k < i + 2; k++) {
                sum = sum - w[k - 1] * A[(i - (k - 1)) * Ny + j];
            }
            for (k = 1; k < l + 1; k++) {
                sum = sum + w[k - 1] * A[(i + k) * Ny + j];
            }
            dA[i * Ny + j] = sum / dx;
        }
        // Between left and right border
        else if (i >= l && i < Nx - l) {
            sum = 0.0;
            for (k = 1; k < l + 1; k++) {
                sum = sum + w[k - 1] * (-A[(i - (k - 1)) * Ny + j] + A[(i + k) * Ny + j]);
            }
            dA[i * Ny + j] = sum / dx;
        }

        
        // Right border
        else {
            sum = 0.0;
            for (k = 1; k < l + 1; k++) {
                sum = sum - w[k - 1] * A[(i - (k - 1)) * Ny + j];
            }
            for (k = 1; k < Nx - i; k++) {
                sum = sum + w[k - 1] * A[(i + k) * Ny + j];
            }
            dA[i * Ny + j] = sum / dx;
        }
    }
}



__global__ void DiffDyplus_kernel(double* A, double* dA, double* w, double dx, int Nx, int Ny, int l) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k;
    double sum;

    if (i < Nx && j < Ny) {
        // Top border (1 < j < l+1)
        if (j < l) {
            sum = 0.0;
            for (k = 1; k < j + 2; k++) {
                sum = -w[k - 1] * A[i * Ny + (j - (k - 1))] + sum;
            }
            for (k = 1; k < l + 1; k++) {
                sum = w[k - 1] * A[i * Ny + (j + k)] + sum;
            }
            dA[i * Ny + j] = sum / dx;
        }
        // Between top and bottom border
        else if (j >= l && j < Ny - l) {
            sum = 0.0;
            for (k = 1; k < l + 1; k++) {
                sum = w[k - 1] * (-A[i * Ny + (j - (k - 1))] + A[i * Ny + (j + k)]) + sum;
            }
            dA[i * Ny + j] = sum / dx;
        }
        // Bottom border
        else if (j >= Ny - l) {
            sum = 0.0;
            for (k = 1; k < l + 1; k++) {
                sum = -w[k - 1] * A[i * Ny + (j - (k - 1))] + sum;
            }
            for (k = 1; k < Ny - j; k++) {
                sum = w[k - 1] * A[i * Ny + (j + k)] + sum;
            }
            dA[i * Ny + j] = sum / dx;
        }
    }
}


__global__ void DiffDxminus_kernel(double* A, double* dA, double* w, double dx, int Nx, int Ny, int l) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k;
    double sum;

    if (i < Nx && j < Ny) {
        // Left border (1 < i < l+1)
        if (i < l) {
            sum = 0.0;
            for (k = 1; k < i + 1; k++) {
                sum = -w[k - 1] * A[(i - k) * Ny + j] + sum;
            }
            for (k = 1; k < l + 1; k++) {
                sum = w[k - 1] * A[(i + (k - 1)) * Ny + j] + sum;
            }
            dA[i * Ny + j] = sum / dx;
        }
        // Outside border area
        else if (i >= l && i < Nx - l) {
            sum = 0.0;
            for (k = 1; k < l + 1; k++) {
                sum = w[k - 1] * (-A[(i - k) * Ny + j] + A[(i + (k - 1)) * Ny + j]) + sum;
            }
            dA[i * Ny + j] = sum / dx;
        }
        // Right border
        else if (i >= Nx - l) {
            sum = 0.0;
            for (k = 1; k < l + 1; k++) {
                sum = -w[k - 1] * A[(i - k) * Ny + j] + sum;
            }
            for (k = 1; k < (Nx - i + 1); k++) {
                sum = w[k - 1] * A[(i + (k - 1)) * Ny + j] + sum;
            }
            dA[i * Ny + j] = sum / dx;
        }
    }
}


__global__ void DiffDyminus_kernel(double* A, double* dA, double* w, double dx, int Nx, int Ny, int l) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k;
    double sum;

    if (i < Nx && j < Ny) {
        // Top border (1 < j < l+1)
        if (j < l) {
            sum = 0.0;
            for (k = 1; k < j + 1; k++) {
                sum = -w[k - 1] * A[i * Ny + (j - k)] + sum;
            }
            for (k = 1; k < l + 1; k++) {
                sum = w[k - 1] * A[i * Ny + (j + (k - 1))] + sum;
            }
            dA[i * Ny + j] = sum / dx;
        }
        // Outside border area
        else if (j >= l && j < Ny - l) {
            sum = 0.0;
            for (k = 1; k < l + 1; k++) {
                sum = w[k - 1] * (-A[i * Ny + (j - k)] + A[i * Ny + (j + (k - 1))]) + sum;
            }
            dA[i * Ny + j] = sum / dx;
        }
        // Bottom border (Ny - l ≤ j < Ny)
        else if (j >= Ny - l) {
            sum = 0.0;
            for (k = 1; k < l + 1; k++) {
                sum = -w[k - 1] * A[i * Ny + (j - k)] + sum;
            }
            for (k = 1; k < (Ny - j + 1); k++) {
                sum = w[k - 1] * A[i * Ny + (j + (k - 1))] + sum;
            }
            dA[i * Ny + j] = sum / dx;
        }
    }
}


__global__ void add_source_kernel(double* p, double Dt, double* Src, double dx, int src_x, int src_y, int Ny, double* Rho, int i) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx == src_x && idy == src_y) {
        p[src_x * Ny + src_y] = p[src_x * Ny + src_y] + Dt * (Src[i] / (dx * dx * Rho[src_x * Ny + src_y]));
    }
}

