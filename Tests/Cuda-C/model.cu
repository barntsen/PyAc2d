#include "hip/hip_runtime.h"
#include "model.h"

//Internal functions
Model* Modelmaxwell(double** vp, double** rho, double** Q, double Dx, double Dt, double W0, int Nb, int Nx, int Ny); // Maxwell Q-model
Model* Modelsls(double** vp, double** rho, double** Q, double Dx, double Dt, double W0, int Nb, int Nx, int Ny); // Sls Q-model
int Modeld(double* d, double dx, int nb, int n);// 1D profile function


Model *ModelNew(double **vp, double **rho, double **Q, double Dx, double Dt, double W0, int Nb, int Rheol, int Nx, int Ny){
    
    Model* m = NULL;

    if(Rheol == MAXWELL){
        m = Modelmaxwell(vp, rho, Q, Dx, Dt, W0, Nb, Nx, Ny); 
    } else if(Rheol == SLS){
        m= Modelsls(vp, rho, Q, Dx, Dt, W0, Nb, Nx, Ny);
    } else{
        fprintf (stderr, "Uknown Q-model\n");
        return NULL;
    } 
    
    return(m);
}


void ModelDel(Model* model, int Nx) {
    int i;
    for (i = 0; i < Nx; i++) {
        free(model->Q[i]);
        free(model->Kappa[i]);
        free(model->Rho[i]);
        
        free(model->Dkappax[i]);
        free(model->Dkappay[i]);
        free(model->Drhox[i]);
        free(model->Drhoy[i]);
        free(model->Alpha1x[i]);
        free(model->Alpha1y[i]);
        free(model->Alpha2x[i]);
        free(model->Alpha2y[i]);
        free(model->Eta1x[i]);
        free(model->Eta1y[i]);
        free(model->Eta2x[i]);
        free(model->Eta2y[i]);
    }
    /*free(model->Nx)
    free(model->Ny)*/
    free(model->dx);
    free(model->dy);

    free(model->Dkappax);
    free(model->Dkappay);
    free(model->Drhox);
    free(model->Drhoy);
    free(model->Alpha1x);
    free(model->Alpha1y);
    free(model->Alpha2x);
    free(model->Alpha2y);
    free(model->Eta1x);
    free(model->Eta1y);
    free(model->Eta2x);
    free(model->Eta2y);

    free(model->Q);
    free(model->Kappa);
    free(model->Rho);
    free(model);

    return;
}


double ModelStability(Model *model){
    
    int i,j;
    double vp,stab;

    for(i=0; i<model->Nx; i=i+1){
        for(j=0; j<model->Ny; j=j+1){
            vp = sqrt(model->Kappa[i][j]/model->Rho[i][j]);
            stab = (vp*model->Dt)/model->Dx;
            if(stab > 1.0/sqrt(2.0)){
                fprintf(stderr,"Stability index too large! %f\n", stab);
            }
        }
    }

    return stab;
}


Model *Modelmaxwell(double **vp, double **rho, double **Q, double Dx, double Dt, double W0, int Nb, int Nx, int Ny){   
    
    int i, j;
    

    // Smoothing parameters
    double Qmin, Qmax;       // Minimum and Maximum Q-values in boundary zone
    double tau0min,tau0max;  // Taue values corresponding to Qmin and Qmax
    double tau0x, tau0y;     // Relaxation times
    double argx, argy;            // Temp variables

    // Allocate memory for the Model struct
    Model* model = (Model*) malloc(sizeof(Model));
    

    // Set the parameters for the Model struct
    model->Dx = Dx; 
    model->Dt = Dt;
    model->W0 = W0;
    model->Nb = Nb;
    model->Nx = Nx;
    model->Ny = Ny;
    
    // Allocate memory for the arrays in the Model struct
    model->dx = (double*) malloc(model->Nx * sizeof(double));
    model->dy = (double*) malloc(model->Ny * sizeof(double));

    // Allocate memory for the double** pointers inside the Model struct
    model->Q = (double**)malloc(model->Nx * sizeof(double*));
    model->Kappa = (double**)malloc(model->Nx * sizeof(double*));
    model->Rho = (double**)malloc(model->Nx * sizeof(double*));

    model->Dkappax = (double**) malloc(model->Nx * sizeof(double*));
    model->Dkappay = (double**) malloc(model->Nx * sizeof(double*));
    model->Drhox = (double**) malloc(model->Nx * sizeof(double*));
    model->Drhoy = (double**) malloc(model->Nx * sizeof(double*));
    model->Alpha1x = (double**) malloc(model->Nx * sizeof(double*));
    model->Alpha1y = (double**) malloc(model->Nx * sizeof(double*));
    model->Alpha2x = (double**) malloc(model->Nx * sizeof(double*));
    model->Alpha2y = (double**) malloc(model->Nx * sizeof(double*));
    model->Eta1x = (double**) malloc(model->Nx * sizeof(double*));
    model->Eta1y = (double**) malloc(model->Nx * sizeof(double*));
    model->Eta2x = (double**) malloc(model->Nx * sizeof(double*));
    model->Eta2y = (double**) malloc(model->Nx * sizeof(double*));
    
    // Allocate memory for the data arrays and assign them to the double** pointers
    for (i = 0; i < model->Nx; i++) {
        model->Q[i] = (double*)malloc(model->Ny * sizeof(double));
        model->Kappa[i] = (double*)malloc(model->Ny * sizeof(double));
        model->Rho[i] = (double*)malloc(model->Ny * sizeof(double));

        model->Dkappax[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Dkappay[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Drhox[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Drhoy[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Alpha1x[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Alpha1y[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Alpha2x[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Alpha2y[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Eta1x[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Eta1y[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Eta2x[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Eta2y[i] = (double*) malloc(model->Ny * sizeof(double));

        for (j = 0; j < model->Ny; j++) {
            model->Kappa[i][j] =  rho[i][j]*pow(vp[i][j], 2);
            model->Q[i][j] = Q[i][j];
            model->Rho[i][j] = rho[i][j];
        }
    }
    
    Modeld(model->dx, model->Dx, model->Nb, model->Nx);
    Modeld(model->dy, model->Dx, model->Nb, model->Ny);
    
    for(i=0; i<model->Nx;i=i+1){
        for(j=0; j<model->Ny;j=j+1){

            // Compute relaxation times corresponding to Qmax and Qmin
            // Note that we compute the inverse of tau0, and use the same name for the inverse, tau0=1/tau0.
            Qmin = 1.1;  // MinimumQ-value at the outer boundaries:    
            tau0min = Qmin/model->W0;
            tau0min = 1.0/tau0min;
            Qmax  = model->Q[Nb][j];
            tau0max = Qmax/model->W0;
            tau0max = 1.0/tau0max;
            // Interpolate tau0 in x-direxction
            tau0x = tau0min + (tau0max-tau0min)*model->dx[i];

            Qmax  = model->Q[i][Nb];
            tau0max = Qmax/model->W0;
            tau0max = 1.0/tau0max;

            // Interpolate tau0 in y-direxction
            tau0y = tau0min + (tau0max-tau0min)*model->dy[j];


            // In the equations below the relaxation time tau0 
            // is inverse (1/tau0)
            // Compute alpha and eta coefficients
            argx = model->dx[i];
            argy = model->dy[j];
            // An extra tapering factor of exp(-(x/L)**2)
            // is used to taper some coefficeints 
            model->Alpha1x[i][j]   = exp(-argx)*exp(-model->Dt*tau0x);
            model->Alpha1y[i][j]   = exp(-argy)*exp(-model->Dt*tau0y);
            model->Alpha2x[i][j]   = -model->Dt*tau0x;
            model->Alpha2y[i][j]   = -model->Dt*tau0y;
            model->Eta1x[i][j]     = exp(-argx)*exp(-model->Dt*tau0x);
            model->Eta1y[i][j]     = exp(-argy)*exp(-model->Dt*tau0y);
            model->Eta2x[i][j]     = -model->Dt*tau0x;
            model->Eta2y[i][j]     = -model->Dt*tau0y;
        
            // For the Maxwell solid Dkappa = kappa and Drho = 1/rho
            // to comply with the solver algorithm i ac2d.e
            model->Dkappax[i][j]   = model->Kappa[i][j];
            model->Dkappay[i][j]   = model->Kappa[i][j];
            model->Drhox[i][j]     = (1.0/model->Rho[i][j]);
            model->Drhoy[i][j]     = (1.0/model->Rho[i][j]);
        }
    } 
    
    return model;
}


Model *Modelsls(double **vp, double **rho, double **Q, double Dx, double Dt, double W0, int Nb, int Nx, int Ny){
        
    int i, j;

    // Smoothing parameters
    double Qmin, Qmax;       // Minimum and Maximum Q-values in boundary zone
    double argx, argy;            // Temp variables
    
    double tau0;         // Relaxation time at Peak 1/Q-value
    double tauemin,tauemax;  // Taue values corresponding to Qmin and Qmax
    double tausmin,tausmax;  // Taus values corresponding to Qmin and Qmax

    // Relaxation times
    double tausx, tausy;     
    double tauex, tauey;


    // Allocate memory for the Model struct
    Model* model = (Model*) malloc(sizeof(Model));

    // Set the parameters for the Model struct
    model->Dx = Dx; 
    model->Dt = Dt;
    model->W0 = W0;
    model->Nb = Nb;
    model->Nx = Nx;
    model->Ny = Ny;
    
    // Allocate memory for the arrays in the Model struct
    model->dx = (double*) malloc(model->Nx * sizeof(double));
    model->dy = (double*) malloc(model->Ny * sizeof(double));

    // Allocate memory for the double** pointers inside the Model struct
    model->Q = (double**)malloc(model->Nx * sizeof(double*));
    model->Kappa = (double**)malloc(model->Nx * sizeof(double*));
    model->Rho = (double**)malloc(model->Nx * sizeof(double*));

    model->Dkappax = (double**) malloc(model->Nx * sizeof(double*));
    model->Dkappay = (double**) malloc(model->Nx * sizeof(double*));
    model->Drhox = (double**) malloc(model->Nx * sizeof(double*));
    model->Drhoy = (double**) malloc(model->Nx * sizeof(double*));
    model->Alpha1x = (double**) malloc(model->Nx * sizeof(double*));
    model->Alpha1y = (double**) malloc(model->Nx * sizeof(double*));
    model->Alpha2x = (double**) malloc(model->Nx * sizeof(double*));
    model->Alpha2y = (double**) malloc(model->Nx * sizeof(double*));
    model->Eta1x = (double**) malloc(model->Nx * sizeof(double*));
    model->Eta1y = (double**) malloc(model->Nx * sizeof(double*));
    model->Eta2x = (double**) malloc(model->Nx * sizeof(double*));
    model->Eta2y = (double**) malloc(model->Nx * sizeof(double*));

    // Allocate memory for the data arrays and assign them to the double** pointers
    for (i = 0; i < model->Nx; i++) {
        model->Q[i] = (double*)malloc(model->Ny * sizeof(double));
        model->Kappa[i] = (double*)malloc(model->Ny * sizeof(double));
        model->Rho[i] = (double*)malloc(model->Ny * sizeof(double));

        model->Dkappax[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Dkappay[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Drhox[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Drhoy[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Alpha1x[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Alpha1y[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Alpha2x[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Alpha2y[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Eta1x[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Eta1y[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Eta2x[i] = (double*) malloc(model->Ny * sizeof(double));
        model->Eta2y[i] = (double*) malloc(model->Ny * sizeof(double));

        for (j = 0; j < model->Ny; j++) {
            model->Kappa[i][j] =  rho[i][j]*pow(vp[i][j], 2);
            model->Q[i][j] = Q[i][j];
            model->Rho[i][j] = rho[i][j];
        }
    }
    
    //Compute 1D profile functions
    Modeld(model->dx, model->Dx, model->Nb, model->Nx);
    Modeld(model->dy, model->Dx, model->Nb, model->Ny);

    // Compute relaxation times
    for(i=0; i<model->Nx;i=i+1){
        for(j=0; j<model->Ny;j=j+1){
            tau0 = 1.0/model->W0;   // Relaxation time corresponding to absorption top
            Qmin = 1.1;            // MinimumQ-value at the outer boundaries

            // Compute relaxation times corresponding to Qmax and Qmin
            tauemin = (tau0/Qmin)*(sqrt(Qmin*Qmin+1.0)+1.0);
            tauemin = 1.0/tauemin;
            tausmin = (tau0/Qmin)*(sqrt(Qmin*Qmin+1.0)-1.0);
            tausmin = 1.0/tausmin;

            Qmax  = model->Q[Nb][j];
            // Note that we compute the inverse
            // of relaxation times, and use the same
            // name for the inverses, taus=1/taus.
            // In all formulas below this section we
            // work with the inverse of the relaxation times.
            tauemax = (tau0/Qmin)*(sqrt(Qmax*Qmax+1.0)+1.0);
            tauemax = 1.0/tauemax;
            tausmax = (tau0/Qmin)*(sqrt(Qmax*Qmax+1.0)-1.0);
            tausmax = 1.0/tausmax;
            tauex = tauemin + (tauemax-tauemin)*model->dx[i];
            tausx = tausmin + (tausmax-tausmin)*model->dx[i];
            Qmax  = model->Q[i][Nb];
            tauemax = (tau0/Qmin)*(sqrt(Qmax*Qmax+1.0)+1.0);
            tauemax = 1.0/tauemax;
            tausmax = (tau0/Qmin)*(sqrt(Qmax*Qmax+1.0)-1.0);
            tausmax = 1.0/tausmax;

            // Interpolate relaxation times 
            tauey = tauemin + (tauemax-tauemin)*model->dy[j];
            tausy = tausmin + (tausmax-tausmin)*model->dy[j];

            // In the equations below the relaxation times taue and taus
            // are inverses (1/taue, 1/taus)
            // Compute alpha and eta coefficients
            argx = model->dx[i];
            argy = model->dy[j];
            // An extra tapering factor of exp(-(x/L)**2)
            // is used to taper some coefficeints 
            model->Alpha1x[i][j]   = exp(-argx)*exp(-model->Dt*tausx);
            model->Alpha1y[i][j]   = exp(-argy)*exp(-model->Dt*tausy);
            model->Alpha2x[i][j]   = model->Dt*tauex;
            model->Alpha2y[i][j]   = model->Dt*tauey;
            model->Eta1x[i][j]     = exp(-argx)*exp(-model->Dt*tausx);
            model->Eta1y[i][j]     = exp(-argy)*exp(-model->Dt*tausy);
            model->Eta2x[i][j]     = model->Dt*tauex;
            model->Eta2y[i][j]     = model->Dt*tauey;
        
            // Compute the change in moduli due to
            // visco-ealsticity (is equal to zero for the elastic case)
            model->Dkappax[i][j]   = model->Kappa[i][j]*(1.0-tausx/tauex);
            model->Dkappay[i][j]   = model->Kappa[i][j]*(1.0-tausy/tauey);
            model->Drhox[i][j]     = (1.0/model->Rho[i][j])*(1.0-tausx/tauex);
            model->Drhoy[i][j]     = (1.0/model->Rho[i][j])*(1.0-tausy/tauey);
        }
    }
    return model;
}


int Modeld(double *d, double dx, int nb, int n){
    
    int i;
    
    for(i=0; i<n; i=i+1){
        d[i]=1.0;
    }
    
    // Taper left border
    for(i=0; i<nb;i=i+1){
        d[i] = d[i]*(((double)i*dx)/((double)nb*dx)
                    *((double)i)*dx/((double)nb*dx));
    }

    // taper right border
    for(i=n-1-nb; i<n;i=i+1){
        d[i] = d[i]*(((double)n-1-i)*dx)/((double)nb*dx)
                    *(((double)n-1-i)*dx)/((double)nb*dx);
    }

    return(OK);
}
