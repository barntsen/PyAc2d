#include "AC2D.h"
#include "diff.h"

#include "CUDA_kernels.cu"



AC2D* Ac2dNew(Model* model){
    
    int i, j;

    AC2D* ac2dini=NULL;
    ac2dini = (AC2D*)malloc(sizeof(AC2D));
    if (!ac2dini){
        fprintf (stderr, "%s\n", strerror(errno));
        return NULL;
    }

    ac2dini->p = (double**) malloc(model->Nx * sizeof(double*));
    ac2dini->vx = (double**) malloc(model->Nx * sizeof(double*));
    ac2dini->vy = (double**) malloc(model->Nx * sizeof(double*));
    ac2dini->exx = (double**) malloc(model->Nx * sizeof(double*));
    ac2dini->eyy = (double**) malloc(model->Nx * sizeof(double*));
    ac2dini->gammax = (double**) malloc(model->Nx * sizeof(double*));
    ac2dini->gammay = (double**) malloc(model->Nx * sizeof(double*));
    ac2dini->thetay = (double**) malloc(model->Nx * sizeof(double*));
    ac2dini->thetax = (double**) malloc(model->Nx * sizeof(double*));

    ac2dini->ts = 0;

    for (i=0; i < model->Nx; i++){ 
        ac2dini->p[i] = (double*) malloc(model->Ny * sizeof(double));
        ac2dini->vx[i] = (double*) malloc(model->Ny * sizeof(double));
        ac2dini->vy[i] = (double*) malloc(model->Ny * sizeof(double));
        ac2dini->exx[i] = (double*) malloc(model->Ny * sizeof(double));
        ac2dini->eyy[i] = (double*) malloc(model->Ny * sizeof(double));
        ac2dini->gammax[i] = (double*) malloc(model->Ny * sizeof(double));
        ac2dini->gammay[i] = (double*) malloc(model->Ny * sizeof(double));
        ac2dini->thetax[i] = (double*) malloc(model->Ny * sizeof(double));
        ac2dini->thetay[i] = (double*) malloc(model->Ny * sizeof(double));
        for (j=0; j < model->Ny; j++){ 
            ac2dini->p[i][j] = 0;
            ac2dini->vx[i][j] = 0;
            ac2dini->vy[i][j] = 0;
            ac2dini->exx[i][j] = 0;
            ac2dini->eyy[i][j] = 0;
            ac2dini->gammax[i][j] = 0;
            ac2dini->gammay[i][j] = 0;
            ac2dini->thetax[i][j] = 0;
            ac2dini->thetay[i][j] = 0;
        }
    }

    return ac2dini;
}


void ac2dDel(AC2D* ac2d, int Nx){

  int i;
    for (i = 0; i < Nx; i++) {
        free(ac2d->p[i]);
        free(ac2d->vx[i]);
        free(ac2d->vy[i]);
        free(ac2d->exx[i]);
        free(ac2d->eyy[i]);
        free(ac2d->gammax[i]);
        free(ac2d->gammay[i]);
        free(ac2d->thetax[i]);
        free(ac2d->thetay[i]);
    }        
    free(ac2d->p);
    free(ac2d->vx);
    free(ac2d->vy);
    free(ac2d->exx);
    free(ac2d->eyy);
    free(ac2d->gammax);
    free(ac2d->gammay);
    free(ac2d->thetax);
    free(ac2d->thetay);

    free(ac2d);
}


int Ac2dSolve(AC2D* ac2d, Model* model, Src* src, Rec* rec,int nt, int l){

    int ns,ne; // Start stop timesteps
    int i;
    double perc, oldperc; // Percentage finished current and old
    int iperc; // Percentage finished

    
    Diff* diff = NULL; // Differentiator object
    diff = DiffNew(l); // Create differentiator object
    oldperc = 0.0;
    ns = ac2d->ts; //Get current timestep 
    ne = ns + nt;        

    // CUDA STARTS HERE

    // 1. Flatten the 2D array to a 1D array
    int Nx = model->Nx;
    int Ny = model->Ny;
    int size = Nx * Ny;
    double* w_flat = (double*) malloc(l * sizeof(double));
    for (int y = 0; y < l; y++) {
        w_flat[y] = diff->w[y]; 
    }

    double* Src_flat = (double*) malloc(nt * sizeof(double));
    for (int z = 0; z < nt; z++) {
        Src_flat[z] = src->Src[z];
    }

    double* p_flat = (double*) malloc(size * sizeof(double));
    double* vx_flat = (double*) malloc(size * sizeof(double));
    double* vy_flat = (double*) malloc(size * sizeof(double));
    double* exx_flat = (double*) malloc(size * sizeof(double));
    double* eyy_flat = (double*) malloc(size * sizeof(double));
    double* gammax_flat = (double*) malloc(size * sizeof(double));
    double* gammay_flat = (double*) malloc(size * sizeof(double));
    double* thetax_flat = (double*) malloc(size * sizeof(double));
    double* thetay_flat = (double*) malloc(size * sizeof(double));
    double* Rho_flat = (double*) malloc(size * sizeof(double));
    double* Kappa_flat = (double*) malloc(size * sizeof(double));
    double* Drhox_flat = (double*) malloc(size * sizeof(double));
    double* Drhoy_flat = (double*) malloc(size * sizeof(double));
    double* Eta1x_flat = (double*) malloc(size * sizeof(double));
    double* Eta2x_flat = (double*) malloc(size * sizeof(double));
    double* Eta1y_flat = (double*) malloc(size * sizeof(double));
    double* Eta2y_flat = (double*) malloc(size * sizeof(double));
    double* Dkappax_flat = (double*) malloc(size * sizeof(double));
    double* Dkappay_flat = (double*) malloc(size * sizeof(double));
    double* Alpha1x_flat = (double*) malloc(size * sizeof(double));
    double* Alpha2x_flat = (double*) malloc(size * sizeof(double));
    double* Alpha1y_flat = (double*) malloc(size * sizeof(double));
    double* Alpha2y_flat = (double*) malloc(size * sizeof(double));

    for (int i = 0; i < Nx; i++) {
        for (int j = 0; j < Ny; j++) {
            p_flat[i * Ny + j] = ac2d->p[i][j];
            vx_flat[i * Ny + j] = ac2d->vx[i][j];
            vy_flat[i * Ny + j] = ac2d->vy[i][j];
            exx_flat[i * Ny + j] = ac2d->exx[i][j];
            eyy_flat[i * Ny + j] = ac2d->eyy[i][j];
            gammax_flat[i * Ny + j] = ac2d->gammax[i][j];
            gammay_flat[i * Ny + j] = ac2d->gammay[i][j];
            thetax_flat[i * Ny + j] = ac2d->thetax[i][j];
            thetay_flat[i * Ny + j] = ac2d->thetay[i][j];
            Rho_flat[i * Ny + j] = model->Rho[i][j];
            Kappa_flat[i * Ny + j] = model->Kappa[i][j];
            Drhox_flat[i * Ny + j] = model->Drhox[i][j];
            Drhoy_flat[i * Ny + j] = model->Drhoy[i][j];
            Eta1x_flat[i * Ny + j] = model->Eta1x[i][j];
            Eta2x_flat[i * Ny + j] = model->Eta2x[i][j];
            Eta1y_flat[i * Ny + j] = model->Eta1y[i][j];
            Eta2y_flat[i * Ny + j] = model->Eta2y[i][j];
            Dkappax_flat[i * Ny + j] = model->Dkappax[i][j];
            Dkappay_flat[i * Ny + j] = model->Dkappay[i][j];
            Alpha1x_flat[i * Ny + j] = model->Alpha1x[i][j];
            Alpha2x_flat[i * Ny + j] = model->Alpha2x[i][j];
            Alpha1y_flat[i * Ny + j] = model->Alpha1y[i][j];
            Alpha2y_flat[i * Ny + j] = model->Alpha2y[i][j];
        }
    }

    // 2. Define the CUDA arrays 
    double* p_gpu, * vx_gpu, * vy_gpu, * exx_gpu, * eyy_gpu, * gammax_gpu, * gammay_gpu, * thetax_gpu, * thetay_gpu; // For ac2d
    double* Rho_gpu, * Kappa_gpu, * Drhox_gpu, * Drhoy_gpu, * Eta1x_gpu, * Eta2x_gpu, * Eta1y_gpu, * Eta2y_gpu, * Dkappax_gpu, * Dkappay_gpu, * Alpha1x_gpu, * Alpha2x_gpu, * Alpha1y_gpu, * Alpha2y_gpu; // For model
    double* w_gpu; // For diff
    double* Src_gpu; // For src

    // 3. Allocate memory on the GPU for the flattened array
    hipMalloc((void**)&p_gpu, size * sizeof(double));
    hipMalloc((void**)&vx_gpu, size * sizeof(double));
    hipMalloc((void**)&vy_gpu, size * sizeof(double));
    hipMalloc((void**)&exx_gpu, size * sizeof(double));
    hipMalloc((void**)&eyy_gpu, size * sizeof(double));
    hipMalloc((void**)&gammax_gpu, size * sizeof(double));
    hipMalloc((void**)&gammay_gpu, size * sizeof(double));
    hipMalloc((void**)&thetax_gpu, size * sizeof(double));
    hipMalloc((void**)&thetay_gpu, size * sizeof(double));
    hipMalloc((void**)&Rho_gpu, size * sizeof(double));
    hipMalloc((void**)&Kappa_gpu, size * sizeof(double));
    hipMalloc((void**)&Drhox_gpu, size * sizeof(double));
    hipMalloc((void**)&Drhoy_gpu, size * sizeof(double));
    hipMalloc((void**)&Eta1x_gpu, size * sizeof(double));
    hipMalloc((void**)&Eta2x_gpu, size * sizeof(double));
    hipMalloc((void**)&Eta1y_gpu, size * sizeof(double));
    hipMalloc((void**)&Eta2y_gpu, size * sizeof(double));
    hipMalloc((void**)&Dkappax_gpu, size * sizeof(double));
    hipMalloc((void**)&Dkappay_gpu, size * sizeof(double));
    hipMalloc((void**)&Alpha1x_gpu, size * sizeof(double));
    hipMalloc((void**)&Alpha2x_gpu, size * sizeof(double));
    hipMalloc((void**)&Alpha1y_gpu, size * sizeof(double));
    hipMalloc((void**)&Alpha2y_gpu, size * sizeof(double));
    hipMalloc((void**)&w_gpu, l * sizeof(double));
    hipMalloc((void**)&Src_gpu, nt * sizeof(double));

    // 4. Copy the flattened array from the host to the GPU
    hipMemcpy(p_gpu, p_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(vx_gpu, vx_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(vy_gpu, vy_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(exx_gpu, exx_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(eyy_gpu, eyy_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gammax_gpu, gammax_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gammay_gpu, gammay_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(thetax_gpu, thetax_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(thetay_gpu, thetay_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Rho_gpu, Rho_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Kappa_gpu, Kappa_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Drhox_gpu, Drhox_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Drhoy_gpu, Drhoy_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Eta1x_gpu, Eta1x_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Eta2x_gpu, Eta2x_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Eta1y_gpu, Eta1y_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Eta2y_gpu, Eta2y_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Dkappax_gpu, Dkappax_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Dkappay_gpu, Dkappay_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Alpha1x_gpu, Alpha1x_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Alpha2x_gpu, Alpha2x_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Alpha1y_gpu, Alpha1y_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Alpha2y_gpu, Alpha2y_flat, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(w_gpu, w_flat, l * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Src_gpu, Src_flat, nt * sizeof(double), hipMemcpyHostToDevice);

    // 4. Perform necessary operations on the GPU
    
    //int blockSize = 1024;
    //int gridSize = (size + blockSize - 1) / blockSize;

    // Determine the block and grid size for a 2D problem
    dim3 blockSize(16, 16); // Adjust these values based on your GPU architecture
    dim3 gridSize((Nx + blockSize.x - 1) / blockSize.x, (Ny + blockSize.y - 1) / blockSize.y);


    for(i=ns; i < ne; i++){

        // Compute spatial derivative of stress
        // Use exx and eyy as temp storage

        // Launch the kernel
        DiffDxplus_kernel<<<gridSize, blockSize>>>(p_gpu, exx_gpu, w_gpu, model->Dx, Nx, Ny, l); // Forward differentiation x-axis
        hipError_t err1 = hipDeviceSynchronize();
        if (err1 != hipSuccess) {
            printf("Error synchronizing the device: %s\n", hipGetErrorString(err1));
            return 1;
        }
        
        // Launch the kernel
        Ac2dvx_kernel<<<gridSize, blockSize>>>(vx_gpu, Rho_gpu, exx_gpu, thetax_gpu, Drhox_gpu, Eta1x_gpu, Eta2x_gpu, Nx, Ny, model->Dt); // Compute vx
        hipError_t err2 = hipDeviceSynchronize();
        if (err2 != hipSuccess) {
            printf("Error synchronizing the device: %s\n", hipGetErrorString(err2));
            return 1;
        }

        // Launch the kernel
        DiffDyplus_kernel<<<gridSize, blockSize>>>(p_gpu, eyy_gpu, w_gpu, model->Dx, Nx, Ny, l); // Forward differentiation y-axis
        hipError_t err3 = hipDeviceSynchronize();
        if (err3 != hipSuccess) {
            printf("Error synchronizing the device: %s\n", hipGetErrorString(err3));
            return 1;
        }

        Ac2dvy_kernel<<<gridSize, blockSize>>>(vy_gpu, Rho_gpu, eyy_gpu, thetay_gpu, Drhoy_gpu, Eta1y_gpu, Eta2y_gpu, Nx, Ny, model->Dt); // Compute vy
        hipError_t err4 = hipDeviceSynchronize();
        if (err4 != hipSuccess) {
            printf("Error synchronizing the device: %s\n", hipGetErrorString(err4));
            return 1;
        }

        // Compute time derivative of strains
        // Launch the kernel
        DiffDxminus_kernel<<<gridSize, blockSize>>>(vx_gpu, exx_gpu, w_gpu, model->Dx, Nx, Ny, l); //Compute exx
        hipError_t err5 = hipDeviceSynchronize();
        if (err5 != hipSuccess) {
            printf("Error synchronizing the device: %s\n", hipGetErrorString(err5));
            return 1;
        }

        // Launch the kernel     
        DiffDyminus_kernel<<<gridSize, blockSize>>>(vy_gpu, eyy_gpu, w_gpu, model->Dx, Nx, Ny, l); //Compute eyy 
        hipError_t err6 = hipDeviceSynchronize();
        if (err6 != hipSuccess) {
            printf("Error synchronizing the device: %s\n", hipGetErrorString(err6));
            return 1;
        }

        // Update stress
        // Launch the kernel
        Ac2dstress_kernel<<<gridSize, blockSize>>>(p_gpu, Kappa_gpu, exx_gpu, eyy_gpu, gammax_gpu, gammay_gpu, Dkappax_gpu, Dkappay_gpu, Alpha1x_gpu, Alpha2x_gpu, Alpha1y_gpu, Alpha2y_gpu, Nx, Ny, model->Dt);
        hipError_t err7 = hipDeviceSynchronize();
        if (err7 != hipSuccess) {
            printf("Error synchronizing the device: %s\n", hipGetErrorString(err7));
            return 1;
        }
        
        // Add source
        add_source_kernel<<<gridSize, blockSize>>>(p_gpu, model->Dt, Src_gpu, model->Dx, src->Sx, src->Sy, Ny, Rho_gpu, i);
        
        // Print progress
        perc=1000.0* (double)i / (double)(ne-ns-1);
        if(perc-oldperc >= 10.0){
            iperc=(int)perc/10;
            if(iperc%10 == 0){printf("%d\n", iperc);}
            oldperc=perc;
        }

        /*
        // 6. Copy the result back from the GPU to the host
        hipMemcpy(p_flat, p_gpu, size * sizeof(double), hipMemcpyDeviceToHost);


        for (int a = 0; a < Nx; a++) {
            for (int b = 0; b < Ny; b++) {
                ac2d->p[a][b] = p_flat[a * Ny + b];
            }
        }
        
        rec_wavefield(rec,i, ac2d->p);
        */
        
    }
   
 
    // 5. Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        exit(1);
    }
    
    
    // 6. Copy the result back from the GPU to the host
    hipMemcpy(p_flat, p_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(vx_flat, vx_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(vy_flat, vy_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(exx_flat, exx_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(eyy_flat, eyy_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(gammax_flat, gammax_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(gammay_flat, gammay_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(thetax_flat, thetax_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(thetay_flat, thetay_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Rho_flat, Rho_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Kappa_flat, Kappa_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Drhox_flat, Drhox_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Drhoy_flat, Drhoy_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Eta1x_flat, Eta1x_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Eta2x_flat, Eta2x_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Eta1y_flat, Eta1y_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Eta2y_flat, Eta2y_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Dkappax_flat, Dkappax_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Dkappay_flat, Dkappay_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Alpha1x_flat, Alpha1x_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Alpha2x_flat, Alpha2x_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Alpha1y_flat, Alpha1y_gpu, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Alpha2y_flat, Alpha2y_gpu, size * sizeof(double), hipMemcpyDeviceToHost);

    // 7. Convert the flattened array back to a 2D array
    for (int i = 0; i < Nx; i++) {
        for (int j = 0; j < Ny; j++) {
            ac2d->p[i][j] = p_flat[i * Ny + j];
            ac2d->vx[i][j] = vx_flat[i * Ny + j];
            ac2d->vy[i][j] = vy_flat[i * Ny + j];
            ac2d->exx[i][j] = exx_flat[i * Ny + j];
            ac2d->eyy[i][j] = eyy_flat[i * Ny + j];
            ac2d->gammax[i][j] = gammax_flat[i * Ny + j];
            ac2d->gammay[i][j] = gammay_flat[i * Ny + j];
            ac2d->thetax[i][j] = thetax_flat[i * Ny + j];
            ac2d->thetay[i][j] = thetay_flat[i * Ny + j];
            model->Rho[i][j] = Rho_flat[i * Ny + j];
            model->Kappa[i][j] = Kappa_flat[i * Ny + j];
            model->Drhox[i][j] = Drhox_flat[i * Ny + j];
            model->Drhoy[i][j] = Drhoy_flat[i * Ny + j];
            model->Eta1x[i][j] = Eta1x_flat[i * Ny + j];
            model->Eta2x[i][j] = Eta2x_flat[i * Ny + j];
            model->Eta1y[i][j] = Eta1y_flat[i * Ny + j];
            model->Eta2y[i][j] = Eta2y_flat[i * Ny + j];
            model->Dkappax[i][j] = Dkappax_flat[i * Ny + j];
            model->Dkappay[i][j] = Dkappay_flat[i * Ny + j];
            model->Alpha1x[i][j] = Alpha1x_flat[i * Ny + j];
            model->Alpha2x[i][j] = Alpha2x_flat[i * Ny + j];
            model->Alpha1y[i][j] = Alpha1y_flat[i * Ny + j];
            model->Alpha2y[i][j] = Alpha2y_flat[i * Ny + j];
        }
    }

    // Free the allocated memory
    free(p_flat);
    free(vx_flat);
    free(vy_flat);
    free(exx_flat);
    free(eyy_flat);
    free(gammax_flat);
    free(gammay_flat);
    free(thetax_flat);
    free(thetay_flat);
    free(Rho_flat);
    free(Kappa_flat);
    free(Drhox_flat);
    free(Drhoy_flat);
    free(Eta1x_flat);
    free(Eta2x_flat);
    free(Eta1y_flat);
    free(Eta2y_flat);
    free(Dkappax_flat);
    free(Dkappay_flat);
    free(Alpha1x_flat);
    free(Alpha2x_flat);
    free(Alpha1y_flat);
    free(Alpha2y_flat);
    free(w_flat);
    free(Src_flat);

    hipFree(p_gpu);
    hipFree(vx_gpu);
    hipFree(vy_gpu);
    hipFree(exx_gpu);
    hipFree(eyy_gpu);
    hipFree(gammax_gpu);
    hipFree(gammay_gpu);
    hipFree(thetax_gpu);
    hipFree(thetay_gpu);
    hipFree(Rho_gpu);
    hipFree(Kappa_gpu);
    hipFree(Drhox_gpu);
    hipFree(Drhoy_gpu);
    hipFree(Eta1x_gpu);
    hipFree(Eta2x_gpu);
    hipFree(Eta1y_gpu);
    hipFree(Eta2y_gpu);
    hipFree(Dkappax_gpu);
    hipFree(Dkappay_gpu);
    hipFree(Alpha1x_gpu);
    hipFree(Alpha2x_gpu);
    hipFree(Alpha1y_gpu);
    hipFree(Alpha2y_gpu);
    hipFree(w_gpu);
    hipFree(Src_gpu);
    

    return OK;
}
