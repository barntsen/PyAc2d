#include "hip/hip_runtime.h"
#include "src.h"


Src *SrcNew(double *source, int sx, int sy){
    Src *src = (Src*)malloc(sizeof(Src));
    src->Src = source;
    src->Sx = sx;
    src->Sy = sy;
    src->Ns = 0; // initialize Ns to 0, assuming you'll update it later
    return src;
}


void SrcDel(Src *src){
    if(src){
        free(src);
        src = NULL;
    }
    return;
}


void SrcRicker(double *src, double t0, double f0, int nt, double dt){
        
    double t, w0, arg;
    int i;
    
    w0 = 2.0*3.1415*f0;
    for(i=0; i<nt; i++){
        t = (double)i*dt-t0;
        arg = w0*t;
        src[i] = (1.0 - 0.5 * pow(arg, 2)) * exp(-0.25 * pow(arg,2));
    }
}
